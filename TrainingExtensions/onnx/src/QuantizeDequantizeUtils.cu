#include "hip/hip_runtime.h"
//==============================================================================
//
//  @@-COPYRIGHT-START-@@
//
//  Copyright (c) 2023, Qualcomm Innovation Center, Inc. All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software
//     without specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
//  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
//  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
//  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
//  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
//  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  @@-COPYRIGHT-END-@@
//
//==============================================================================

#include "QuantizeDequantizeUtils.hpp"


const int CUDA_NUM_THREADS = 512;

// Compute the number of blocks based on the total number of threads.
inline int CUDA_NUM_BLOCKS(const int N)
{
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}


template <typename DTYPE>
__global__ void permuteTensorKernel(const DTYPE* in, DTYPE* out, int numElements, int numDims,
                                    const int64_t* inputStrides, const int64_t* outputStrides)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < numElements; i += blockDim.x * gridDim.x)
    {
        size_t outputIdx = 0;
        size_t remainder = i;
        for (auto dim = 0; dim < numDims; dim++)
        {
            size_t dimIdx = remainder / inputStrides[dim];
            remainder = remainder - dimIdx * inputStrides[dim];
            outputIdx += outputStrides[dim] * dimIdx;
        }

        out[outputIdx] = in[i];
    }
}


template <typename T>
void permuteTensorGPU(const T* inTensor, T* outTensor, int64_t numel, int64_t numDims, const int64_t* inputStrides,
                      const int64_t* outputStrides)
{
    int64_t totalThreads = numel;
    int64_t gridSize     = CUDA_NUM_BLOCKS(totalThreads);
    int64_t strideData[2][numDims];

    // Copy the stride information to the cuda device
    for (int i = 0; i < numDims; i++)
    {
        strideData[0][i] = inputStrides[i];
        strideData[1][i] = outputStrides[i];
    }
    int64_t* deviceStrideData;
    hipMalloc((void**) &deviceStrideData, 2 * numDims * sizeof(int64_t));
    hipMemcpy(deviceStrideData, strideData, 2 * numDims * sizeof(int64_t), hipMemcpyHostToDevice);

    // Launch the cuda kernel
    permuteTensorKernel<<<gridSize, CUDA_NUM_THREADS>>>(inTensor, outTensor, numel, numDims, deviceStrideData,
                                                        deviceStrideData + numDims);

    // Free the device stride data
    hipFree(deviceStrideData);
}


template void permuteTensorGPU(const float* intensor, float* outTensor, int64_t numel, int64_t numDims,
                               const int64_t* inputStrides, const int64_t* outputStrides);